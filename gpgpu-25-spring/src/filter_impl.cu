#include "hip/hip_runtime.h"
#include "filter_impl.h"

#include <cassert>
#include <cstdint>
#include <cstdio>

#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)
template <typename T>
void check(T err, const char* const func, const char* const file,
           const int line)
{
    if (err != hipSuccess)
    {
        std::fprintf(stderr, "CUDA Runtime Error at: %s: %d\n", file, line);
        std::fprintf(stderr, "%s %s\n", hipGetErrorString(err), func);
        // We don't exit when we encounter CUDA errors in this example.
        std::exit(EXIT_FAILURE);
    }
}

struct rgb {
    uint8_t r, g, b;
};

struct rgbState {
    uint8_t r, g, b, time;
};

__device__ float color_distance(const rgb* p1, const rgbState* p2) {
    return sqrtf((p1->r - p2->r) * (p1->r - p2->r) +
                 (p1->g - p2->g) * (p1->g - p2->g) +
                 (p1->b - p2->b) * (p1->b - p2->b));
}

__global__ void motion_first_frame(rgb* dbuffer_frame, size_t pitch_dbuffer_frame, rgbState* dbuffer_background, size_t pitch_dbuffer_background, int width, int height)
{
    int y = blockIdx.y * blockDim.y + threadIdx.y; 
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    if (x >= width || y >= height)
        return;
    rgb* idxFrame = ((rgb*)((char *)dbuffer_frame + y * pitch_dbuffer_frame));
    rgbState* idxBackground = (rgbState*)((char *)dbuffer_background + y * pitch_dbuffer_background);
    idxBackground[x].r = idxFrame[x].r;
    idxBackground[x].g = idxFrame[x].g;
    idxBackground[x].b = idxFrame[x].b;
    idxBackground[x].time = 0;
}

__global__ void motion_detect(rgb* dbuffer_frame, size_t pitch_dbuffer_frame, rgbState* dbuffer_background, size_t pitch_dbuffer_background, uint8_t* dbuffer_grayscale, size_t pitch_dbuffer_grayscale, int width, int height)
{
    int y = blockIdx.y * blockDim.y + threadIdx.y; 
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    if (x >= width || y >= height)
        return;

    rgb* idxFrame = ((rgb*)((char *)dbuffer_frame + y * pitch_dbuffer_frame));
    rgbState* idxBackground = (rgbState*)((char *)dbuffer_background + y * pitch_dbuffer_background);
    float distance = color_distance(&idxFrame[x], &idxBackground[x]);

    if (distance <= 25.0f)
    {
        idxBackground[x].time = 0;
        dbuffer_grayscale[x + y * pitch_dbuffer_grayscale] = (uint8_t)distance;
    }
    else
    {
        dbuffer_grayscale[x + y * pitch_dbuffer_grayscale] = (uint8_t)distance;
        idxBackground[x].time++;
        if (idxBackground[x].time > 5)
        {
            idxBackground[x].r = idxFrame[x].r;
            idxBackground[x].g = idxFrame[x].g;
            idxBackground[x].b = idxFrame[x].b;
            idxBackground[x].time = 0;
            dbuffer_grayscale[x + y * pitch_dbuffer_grayscale] = (uint8_t)distance;
        }
    }
}

__global__ void erosion_row_major(uint8_t* dsrc_grayscale, size_t dsrc_grayscale_pitch, uint8_t* dgrayscale_column_major, size_t dgrayscale_column_major_pitch, int width, int height)
{
    int y = blockIdx.y * blockDim.y + threadIdx.y; 
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    if (x >= width-1 || y >= height-1 || x <= 0 || y <= 0)
        return;

    uint8_t eroded = min(min(dsrc_grayscale[y * dsrc_grayscale_pitch + x-1], 
                             dsrc_grayscale[y * dsrc_grayscale_pitch + x]), 
                         dsrc_grayscale[y * dsrc_grayscale_pitch + x+1]);
    dgrayscale_column_major[x * dgrayscale_column_major_pitch + y] = eroded;
}


__global__ void erosion_column_major(uint8_t* dsrc_grayscale_column_major, size_t dsrc_grayscale_column_major_pitch, uint8_t* dgrayscale_row_major, size_t dgrayscale_row_major_pitch, int width, int height)
{
    int y = blockIdx.y * blockDim.y + threadIdx.y; 
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    if (x >= width-1 || y >= height-1 || x <= 0 || y <= 0)
        return;

    uint8_t eroded = min(min(dsrc_grayscale_column_major[x * dsrc_grayscale_column_major_pitch + y-1], 
                             dsrc_grayscale_column_major[x * dsrc_grayscale_column_major_pitch + y]), 
                         dsrc_grayscale_column_major[x * dsrc_grayscale_column_major_pitch + y+1]);
    dgrayscale_row_major[y * dgrayscale_row_major_pitch + x] = eroded;
}

__global__ void dilation_row_major(uint8_t* dsrc_grayscale, size_t dsrc_grayscale_pitch, uint8_t* dgrayscale_column_major, size_t dgrayscale_column_major_pitch, int width, int height)
{
    int y = blockIdx.y * blockDim.y + threadIdx.y; 
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    if (x >= width-1 || y >= height-1 || x <= 0 || y <= 0)
        return;

    uint8_t dilated = max(max(dsrc_grayscale[y * dsrc_grayscale_pitch + x-1],
                              dsrc_grayscale[y * dsrc_grayscale_pitch + x]), 
                          dsrc_grayscale[y * dsrc_grayscale_pitch + x+1]);
    dgrayscale_column_major[x * dgrayscale_column_major_pitch + y] = dilated;
}

__global__ void dilation_column_major(uint8_t* dsrc_grayscale_column_major, size_t dsrc_grayscale_column_major_pitch, uint8_t* dgrayscale_row_major, size_t dgrayscale_row_major_pitch, int width, int height)
{
    int y = blockIdx.y * blockDim.y + threadIdx.y; 
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    if (x >= width-1 || y >= height-1 || x <= 0 || y <= 0)
        return;

    uint8_t dilated = max(max(dsrc_grayscale_column_major[x * dsrc_grayscale_column_major_pitch + y-1], 
                              dsrc_grayscale_column_major[x * dsrc_grayscale_column_major_pitch + y]), 
                          dsrc_grayscale_column_major[x * dsrc_grayscale_column_major_pitch + y+1]);
    dgrayscale_row_major[y * dgrayscale_row_major_pitch + x] = dilated;
}

__global__ void hysterisis(uint8_t* dbuffer_row_major, size_t pitch_dbuffer_row_major, int width, int height)
{
    int y = blockIdx.y * blockDim.y + threadIdx.y; 
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    if (x >= width-1 || y >= height-1 || x <= 0 || y <= 0)
        return;
    
    if (dbuffer_row_major[x + y * pitch_dbuffer_row_major] >= 30)
        dbuffer_row_major[x + y * pitch_dbuffer_row_major] = 255;
    else if (dbuffer_row_major[x + y * pitch_dbuffer_row_major] < 4)
        dbuffer_row_major[x + y * pitch_dbuffer_row_major] = 0;
    else
    {
        int i = -1;
        int j = -1;
        while (j <= 1)
        {
            while (i >= 1)
            {
                if (dbuffer_row_major[(x + i) + (y + j) * pitch_dbuffer_row_major] >= 30)
                {
                    dbuffer_row_major[x + y * pitch_dbuffer_row_major] = 255;
                    return;
                }
                i++;
            }
            j++;
        }
        dbuffer_row_major[x + y * pitch_dbuffer_row_major] = 0;
    }
}

__global__ void apply_red(rgb* dbuffer_frame, size_t pitch_dbuffer_frame, uint8_t* dbuffer_row_major, size_t pitch_dbuffer_row_major, int width, int height)
{
    int y = blockIdx.y * blockDim.y + threadIdx.y; 
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    if (x >= width || y >= height)
        return;
    

    rgb* idxFrame = ((rgb*)((char *)dbuffer_frame + y * pitch_dbuffer_frame));
    if (dbuffer_row_major[x + y * pitch_dbuffer_row_major] > 0)
        idxFrame[x].r = min(255, (idxFrame[x].r + (uint8_t)(0.5f * 255)));
}

extern "C" {
static uint8_t* dBuffer = nullptr;
static uint8_t* dbuffer_greyscale_row_major = nullptr;
static uint8_t* dbuffer_greyscale_column_major = nullptr;
static rgbState* dbuffer_background = nullptr;

static size_t pitch = 0;
static size_t pitch_dbuffer_greyscale_row_major = 0;
static size_t pitch_dbuffer_greyscale_column_major = 0;
static size_t pitch_dbuffer_background = 0;
static int current_width = 0;
static int current_height = 0;

static hipStream_t stream = nullptr;

void filter_init() {
    hipError_t err = hipStreamCreate(&stream);
    CHECK_CUDA_ERROR(err);
}

void filter_impl(uint8_t* src_buffer, int width, int height, int src_stride, int pixel_stride) {
    assert(sizeof(rgb) == pixel_stride);
    hipError_t err;

    if (!stream) {
        filter_init();
    }

    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, 
                (height + blockSize.y - 1) / blockSize.y);

    if (!dBuffer || width != current_width || height != current_height) {
        if (dBuffer) {
            err = hipFree(dBuffer);
            CHECK_CUDA_ERROR(err);
        }
        if (dbuffer_greyscale_row_major) {
            err = hipFree(dbuffer_greyscale_row_major);
            CHECK_CUDA_ERROR(err);
        }
        if (dbuffer_greyscale_column_major) {
            err = hipFree(dbuffer_greyscale_column_major);
            CHECK_CUDA_ERROR(err);
        }
        if (dbuffer_background) {
            err = hipFree(dbuffer_background);
            CHECK_CUDA_ERROR(err);
        }

        err = hipMallocPitch(&dBuffer, &pitch, width * sizeof(rgb), height);
        CHECK_CUDA_ERROR(err);
        err = hipMallocPitch(&dbuffer_greyscale_row_major, &pitch_dbuffer_greyscale_row_major, 
                             width * sizeof(uint8_t), height);
        CHECK_CUDA_ERROR(err);
        err = hipMallocPitch(&dbuffer_greyscale_column_major, &pitch_dbuffer_greyscale_column_major,
                             height * sizeof(uint8_t), width);
        CHECK_CUDA_ERROR(err);

        err = hipMallocPitch(&dbuffer_background, &pitch_dbuffer_background, width * sizeof(rgbState), height);
        CHECK_CUDA_ERROR(err);
       
        err = hipMemcpy2DAsync(dBuffer, pitch, src_buffer, src_stride, 
                                width * sizeof(rgb), height, hipMemcpyHostToDevice, stream);
        CHECK_CUDA_ERROR(err);
        
        motion_first_frame<<<gridSize, blockSize, 0, stream>>>((rgb*)dBuffer, pitch, dbuffer_background, pitch_dbuffer_background, width, height);

        current_width = width;
        current_height = height;
    }
    else
{

        err = hipMemcpy2DAsync(dBuffer, pitch, src_buffer, src_stride, 
                                width * sizeof(rgb), height, hipMemcpyHostToDevice, stream);
        CHECK_CUDA_ERROR(err);

        motion_detect<<<gridSize, blockSize, 0, stream>>>((rgb*)dBuffer, pitch, dbuffer_background, pitch_dbuffer_background, dbuffer_greyscale_row_major, pitch_dbuffer_greyscale_row_major, width, height);

        erosion_row_major<<<(256, 1), blockSize, 0, stream>>>(
            dbuffer_greyscale_row_major, pitch_dbuffer_greyscale_row_major,
            dbuffer_greyscale_column_major,
            pitch_dbuffer_greyscale_column_major, width, height);

        erosion_column_major<<<(256, 1), blockSize, 0, stream>>>(
            dbuffer_greyscale_column_major, pitch_dbuffer_greyscale_column_major,
            dbuffer_greyscale_row_major, pitch_dbuffer_greyscale_row_major,
            width, height);

        dilation_row_major<<<(256, 1), blockSize, 0, stream>>>(
            dbuffer_greyscale_row_major, pitch_dbuffer_greyscale_row_major,
            dbuffer_greyscale_column_major, pitch_dbuffer_greyscale_column_major,
            width, height);

        dilation_column_major<<<(256, 1), blockSize, 0, stream>>>(
            dbuffer_greyscale_column_major, pitch_dbuffer_greyscale_column_major,
            dbuffer_greyscale_row_major, pitch_dbuffer_greyscale_row_major,
            width, height);

        hysterisis<<<gridSize, blockSize, 0, stream>>>(dbuffer_greyscale_row_major, pitch_dbuffer_greyscale_row_major, width, height);

        apply_red<<<gridSize, blockSize, 0, stream>>>((rgb*)dBuffer, pitch, dbuffer_greyscale_row_major, pitch_dbuffer_greyscale_row_major, width, height);
    }

    err = hipMemcpy2DAsync(src_buffer, src_stride, dBuffer, pitch, 
                            width * sizeof(rgb), height, hipMemcpyDeviceToHost, stream);
    CHECK_CUDA_ERROR(err);

    err = hipStreamSynchronize(stream);
    CHECK_CUDA_ERROR(err);
}

void filter_cleanup() {
    if (dBuffer) hipFree(dBuffer);
    if (dbuffer_greyscale_row_major) hipFree(dbuffer_greyscale_row_major);
    if (dbuffer_greyscale_column_major) hipFree(dbuffer_greyscale_column_major);
    
    if (stream) {
        hipStreamDestroy(stream);
        stream = nullptr;
    }

    dBuffer = nullptr;
    dbuffer_greyscale_row_major = nullptr;
    dbuffer_greyscale_column_major = nullptr;
    current_width = 0;
    current_height = 0;
}
}
